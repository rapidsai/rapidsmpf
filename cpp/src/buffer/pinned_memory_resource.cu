/**
 * SPDX-FileCopyrightText: Copyright (c) 2024-2025, NVIDIA CORPORATION & AFFILIATES.
 * SPDX-License-Identifier: Apache-2.0
 */
#include <cstddef>
#include <memory>

#include <hip/hip_runtime_api.h>

#include <cuda/memory_resource>

#include <rmm/resource_ref.hpp>

#include <rapidsmpf/buffer/pinned_memory_resource.hpp>
#include <rapidsmpf/error.hpp>
#include <rapidsmpf/utils.hpp>

#if RAPIDSMPF_CUDA_VERSION_AT_LEAST(RAPIDSMPF_PINNED_MEM_RES_MIN_CUDA_VERSION)
#include <cuda/experimental/memory_resource.cuh>
#endif

namespace rapidsmpf {
#if RAPIDSMPF_CUDA_VERSION_AT_LEAST(RAPIDSMPF_PINNED_MEM_RES_MIN_CUDA_VERSION)
namespace {
cuda::experimental::memory_pool_properties get_memory_pool_properties(
    PinnedPoolProperties const&
) {
    return cuda::experimental::memory_pool_properties{};
}
}  // namespace

// PinnedMemoryPool implementation
struct PinnedMemoryPool::PinnedMemoryPoolImpl {
    PinnedMemoryPoolImpl(int numa_id, PinnedPoolProperties const& properties)
        : p_pool{numa_id, get_memory_pool_properties(properties)} {}

    cuda::experimental::pinned_memory_pool p_pool;
};

// PinnedMemoryResource implementation
struct PinnedMemoryResource::PinnedMemoryResourceImpl {
    PinnedMemoryResourceImpl(PinnedMemoryPool& pool) : p_resource{pool.impl_->p_pool} {}

    void* allocate(rmm::cuda_stream_view stream, size_t bytes) {
        return p_resource.allocate(stream, bytes);
    }

    void* allocate(rmm::cuda_stream_view stream, size_t bytes, size_t alignment) {
        return p_resource.allocate(stream, bytes, alignment);
    }

    void deallocate(rmm::cuda_stream_view stream, void* ptr, size_t bytes) {
        p_resource.deallocate(stream, ptr, bytes);
    }

    void deallocate(
        rmm::cuda_stream_view stream, void* ptr, size_t bytes, size_t alignment
    ) {
        p_resource.deallocate(stream, ptr, bytes, alignment);
    }

    void* allocate_sync(size_t bytes, size_t alignment) {
        return p_resource.allocate_sync(bytes, alignment);
    }

    void deallocate_sync(void* ptr, size_t bytes, size_t alignment) {
        p_resource.deallocate_sync(ptr, bytes, alignment);
    }

    cuda::experimental::pinned_memory_resource p_resource;
};
#else  // CUDA_VERSION < RAPIDSMPF_PINNED_MEM_RES_MIN_CUDA_VERSION
struct PinnedMemoryPool::PinnedMemoryPoolImpl {
    PinnedMemoryPoolImpl(int, PinnedPoolProperties const&) {
        RAPIDSMPF_FAIL(
            "PinnedMemoryPool is not supported for CUDA versions "
            "below " RAPIDSMPF_PINNED_MEM_RES_MIN_CUDA_VERSION_STR
        );
    }
};

struct PinnedMemoryResource::PinnedMemoryResourceImpl {
    PinnedMemoryResourceImpl(PinnedMemoryPool&) {
        RAPIDSMPF_FAIL(
            "PinnedMemoryResource is not supported for CUDA versions "
            "below " RAPIDSMPF_PINNED_MEM_RES_MIN_CUDA_VERSION_STR
        );
    }

    void* allocate(rmm::cuda_stream_view, size_t) {
        return nullptr;
    }

    void* allocate(rmm::cuda_stream_view, size_t, size_t) {
        return nullptr;
    }

    void deallocate(rmm::cuda_stream_view, void*, size_t) {}

    void deallocate(rmm::cuda_stream_view, void*, size_t, size_t) {}

    void* allocate_sync(size_t, size_t) {}

    void deallocate_sync(void*, size_t, size_t) {}
};
#endif

PinnedMemoryPool::PinnedMemoryPool(
    std::optional<int> numa_id, PinnedPoolProperties properties
)
    : numa_id_(numa_id ? *numa_id : get_current_numa_node_id()),
      properties_(std::move(properties)),
      impl_(std::make_unique<PinnedMemoryPoolImpl>(numa_id_, properties_)) {
    RAPIDSMPF_EXPECTS(
        is_pinned_memory_resources_supported(),
        "PinnedMemoryPool is not supported for CUDA versions "
        "below " RAPIDSMPF_PINNED_MEM_RES_MIN_CUDA_VERSION_STR
    );
}

PinnedMemoryPool::~PinnedMemoryPool() = default;

PinnedMemoryResource::PinnedMemoryResource(PinnedMemoryPool& pool)
    : impl_(std::make_unique<PinnedMemoryResourceImpl>(pool)) {
    RAPIDSMPF_EXPECTS(
        is_pinned_memory_resources_supported(),
        "PinnedMemoryResource is not supported for CUDA versions "
        "below " RAPIDSMPF_PINNED_MEM_RES_MIN_CUDA_VERSION_STR
    );
}

PinnedMemoryResource::~PinnedMemoryResource() = default;

void* PinnedMemoryResource::allocate(rmm::cuda_stream_view stream, size_t bytes) {
    return impl_->allocate(stream, bytes);
}

void* PinnedMemoryResource::allocate(
    rmm::cuda_stream_view stream, size_t bytes, size_t alignment
) {
    return impl_->allocate(stream, bytes, alignment);
}

void PinnedMemoryResource::deallocate(
    rmm::cuda_stream_view stream, void* ptr, size_t bytes
) noexcept {
    impl_->deallocate(stream, ptr, bytes);
}

void PinnedMemoryResource::deallocate(
    rmm::cuda_stream_view stream, void* ptr, size_t bytes, size_t alignment
) noexcept {
    impl_->deallocate(stream, ptr, bytes, alignment);
}

void* PinnedMemoryResource::allocate_sync(size_t bytes, size_t alignment) {
    return impl_->allocate_sync(bytes, alignment);
}

void PinnedMemoryResource::deallocate_sync(void* ptr, size_t bytes, size_t alignment) {
    impl_->deallocate_sync(ptr, bytes, alignment);
}

// PinnedHostBuffer implementation
PinnedHostBuffer::PinnedHostBuffer(
    size_t size, rmm::cuda_stream_view stream, std::shared_ptr<PinnedMemoryResource> mr
)
    : size_(size), stream_(stream), mr_(std::move(mr)) {
    RAPIDSMPF_EXPECTS(mr_ != nullptr, "mr cannot be nullptr", std::invalid_argument);
    data_ = static_cast<std::byte*>(mr_->allocate(stream, size));
}

PinnedHostBuffer::PinnedHostBuffer(
    void const* src_data,
    size_t size,
    rmm::cuda_stream_view stream,
    std::shared_ptr<PinnedMemoryResource> mr
)
    : PinnedHostBuffer(size, stream, std::move(mr)) {
    if (size > 0) {
        RAPIDSMPF_EXPECTS(
            nullptr != src_data, "Invalid copy from nullptr.", std::invalid_argument
        );
        RAPIDSMPF_EXPECTS(
            nullptr != data_, "Invalid copy to nullptr.", std::invalid_argument
        );
        RAPIDSMPF_CUDA_TRY(
            hipMemcpyAsync(data_, src_data, size, hipMemcpyDefault, stream.value())
        );
    }
}

PinnedHostBuffer::~PinnedHostBuffer() noexcept {
    deallocate_async();
}

PinnedHostBuffer::PinnedHostBuffer(PinnedHostBuffer&& other)
    : data_(other.data_),
      size_(other.size_),
      stream_(other.stream_),
      mr_(std::move(other.mr_)) {
    other.data_ = nullptr;
    other.size_ = 0;
}

PinnedHostBuffer& PinnedHostBuffer::operator=(PinnedHostBuffer&& other) {
    if (this != &other) {
        deallocate_async();
        data_ = std::exchange(other.data_, nullptr);
        size_ = std::exchange(other.size_, 0);
        stream_ = other.stream_;
        mr_ = std::move(other.mr_);
    }
    return *this;
}

void PinnedHostBuffer::deallocate_async() noexcept {
    if (mr_ && data_) {
        mr_->deallocate(stream_, data_, size_);
        data_ = nullptr;
        size_ = 0;
    }
}

void PinnedHostBuffer::synchronize() {
    stream_.synchronize();
}

}  // namespace rapidsmpf
