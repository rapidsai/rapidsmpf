/**
 * SPDX-FileCopyrightText: Copyright (c) 2024-2025, NVIDIA CORPORATION & AFFILIATES.
 * SPDX-License-Identifier: Apache-2.0
 */

#include <hip/hip_runtime_api.h>

#include <cuda/experimental/memory_resource.cuh>

#include <rapidsmpf/buffer/pinned_memory_resource.hpp>
#include <rapidsmpf/error.hpp>

namespace rapidsmpf {

namespace {
cuda::experimental::memory_pool_properties get_memory_pool_properties(
    PinnedPoolProperties const& properties
) {
    return cuda::experimental::memory_pool_properties{
        .initial_pool_size = properties.initial_pool_size,
        .release_threshold = properties.release_threshold
    };
}
}  // namespace

// PinnedMemoryPool implementation
struct PinnedMemoryPool::PinnedMemoryPoolImpl {
    PinnedMemoryPoolImpl(
        int numa_id, cuda::experimental::memory_pool_properties properties
    )
        : numa_id(numa_id), p_pool{numa_id, std::move(properties)} {}

    int numa_id;
    cuda::experimental::pinned_memory_pool p_pool;
};

PinnedMemoryPool::PinnedMemoryPool(int numa_id, PinnedPoolProperties properties)
    : numa_id_(numa_id),
      properties_(std::move(properties)),
      impl_(std::make_unique<PinnedMemoryPoolImpl>(
          numa_id, get_memory_pool_properties(properties_)
      )) {}

PinnedMemoryPool::~PinnedMemoryPool() = default;

// PinnedMemoryResource implementation
struct PinnedMemoryResource::PinnedMemoryResourceImpl {
    PinnedMemoryResourceImpl(cuda::experimental::pinned_memory_pool& pool)
        : p_resource{pool} {}

    void* allocate_async(size_t bytes, const cuda::stream_ref stream_ref) {
        return p_resource.allocate_async(bytes, stream_ref);
    }

    void deallocate_async(void* ptr, const cuda::stream_ref stream_ref) {
        p_resource.deallocate_async(ptr, size_t{}, stream_ref);
    }

    cuda::experimental::pinned_memory_resource p_resource;
};

PinnedMemoryResource::PinnedMemoryResource(PinnedMemoryPool& pool)
    : impl_(std::make_unique<PinnedMemoryResourceImpl>(
          pool.impl_->p_pool
      )) {}

PinnedMemoryResource::~PinnedMemoryResource() = default;

void* PinnedMemoryResource::allocate_async(
    size_t bytes, const cuda::stream_ref stream_ref
) {
    return impl_->allocate_async(bytes, stream_ref);
}

void PinnedMemoryResource::deallocate_async(
    void* ptr, const cuda::stream_ref stream_ref
) {
    impl_->deallocate_async(ptr, stream_ref);
}

// PinnedHostBuffer implementation
PinnedHostBuffer::PinnedHostBuffer(
    size_t size, cuda::stream_ref stream, PinnedMemoryResource* p_resource
)
    : size_(size), stream_ref_(stream), p_resource_(p_resource) {
    RAPIDSMPF_EXPECTS(p_resource_ != nullptr, "p_resource cannot be nullptr");
    data_ = p_resource_->allocate_async(size, stream);
}

PinnedHostBuffer::PinnedHostBuffer(
    void const* data,
    size_t size,
    cuda::stream_ref stream,
    PinnedMemoryResource* p_resource
)
    : PinnedHostBuffer(size, stream, p_resource) {
    if (size > 0) {
        RAPIDSMPF_EXPECTS(nullptr != data, "Invalid copy from nullptr.");
        RAPIDSMPF_EXPECTS(nullptr != data_, "Invalid copy to nullptr.");
        RAPIDSMPF_CUDA_TRY(
            hipMemcpyAsync(data_, data, size, hipMemcpyDefault, stream.get())
        );
    }
}

PinnedHostBuffer::~PinnedHostBuffer() noexcept {
    deallocate_async();
    stream_ref_.wait();
}

void PinnedHostBuffer::deallocate_async() noexcept {
    if (p_resource_ && data_) {
        p_resource_->impl_->deallocate_async(data_, stream_ref_);
        data_ = nullptr;
        size_ = 0;
    }
}

}  // namespace rapidsmpf
